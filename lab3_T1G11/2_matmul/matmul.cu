#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define BLOCKSIZE 16

// CUDA ERROR CHECK
#define CUDA_CHECK(call)                                          \
    do {                                                          \
        hipError_t err = (call);                                 \
        if (err != hipSuccess)                                   \
        {                                                         \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",          \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(1);                                              \
        }                                                         \
    } while (0);


// TODO CHECK
// Sequential Matrix Multiplication
void matmul_seq(double *A, double *B, double *C, const int N)
{
	int val = 0;
	int i, j, k;

	for(i = 0; i < N; i++) { // per cada fila de A
		for(j = 0; j < N; j++) { //per cada columna de B
			for(k = 0; k < N; k++) {	// sumar cada element de la fila de A per el mateix element de la columna de B
				val += A[i*N + k] * B[k*N +j];
			}
			C[i*N + j] = val;
			val = 0;
		}
	}
}


// TODO CHECK
// Simple CUDA Matrix Multiplication Kernel
__global__ void matmul_naive_kernel(double *A, double *B, double *C, const int N)
{
	// claculem coordenades de C d'aquest thread
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	// si el thread està fora de la matriu retorna
	if(row >= N || col >= N) return;

	// claculem el valor de A[row] · B[col]
	int val = 0;
	for(int i = 0; i < N; i++) {
		val += A[row*N + i] * B[i*N + col];
	}

	// assigna el valor a C
	C[row*N + col] = val;
}


// TODO CHECK
// Matrix Multiplication Kernel exploiting shared memory
__global__ void matmul_shared_kernel(double *A, double *B, double *C, const int N)
{
	// crear les tiles
	__shared__ double A_tile[BLOCKSIZE][BLOCKSIZE];
	__shared__ double B_tile[BLOCKSIZE][BLOCKSIZE];

	// BLOCKSIZE és el mateix que blockDim perquè hemm fet les tiles de la mida dels blocks
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;

	int i, k;
	int val = 0;

	if(row >= N || col >= N) return;

	for(i = 0; i < (BLOCKSIZE + N -1)/BLOCKSIZE; i++) {
		if(i * BLOCKSIZE + threadIdx.x < N && row < N)
			A_tile[threadIdx.y][threadIdx.x] = A[row*N + i*BLOCKSIZE + threadIdx.x];
		else
			A_tile[threadIdx.y][threadIdx.x] = 0.0;

		if(i * BLOCKSIZE + threadIdx.y < N && col < N)
			B_tile[threadIdx.x][threadIdx.y] = B[col*N + i*BLOCKSIZE + threadIdx.y];
		else
			B_tile[threadIdx.x][threadIdx.y] = 0.0;

		__syncthreads();

		for(k = 0; k < N; k++)
			val += A_tile[threadIdx.y][k]*B_tile[k][threadIdx.x];

		__syncthreads();
	}

	if(row < N && col < N)
		C[row*N + col] = val;

}


void validation(double *h_C, double *C, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            double err = fabs(h_C[i * N + j] - C[i * N + j]);
            if (err > 1.0e-6)
            {
                printf("Error at C[%d][%d]: fabs( %f - %f ) = %e > %e\n", i, j, h_C[i * N + j], C[i * N + j], err, 1.0e-6);
                exit(1);
            }
        }
    }
}

void copy_A_B_H2D(double *h_A, double *h_B, double *d_A, double *d_B, const size_t bytes,
                  hipEvent_t *event_start, hipEvent_t *event_end, float *total_time_ms, const char *case_name)
{
    float time_ms = 0.0;
    CUDA_CHECK(hipEventRecord(*event_start));

    CUDA_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    CUDA_CHECK(hipEventRecord(*event_end));
    CUDA_CHECK(hipEventSynchronize(*event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, *event_start, *event_end));
    printf("%s GPU H2D copy time: %.9f seconds\n", case_name, time_ms / 1000);
    *total_time_ms += time_ms;
}

void copy_C_D2H(double *h_C, double *d_C, const size_t bytes,
                hipEvent_t *event_start, hipEvent_t *event_end, float *total_time_ms, const char *case_name)
{
    float time_ms = 0.0;
    CUDA_CHECK(hipEventRecord(*event_start));

    CUDA_CHECK(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipEventRecord(*event_end));
    CUDA_CHECK(hipEventSynchronize(*event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, *event_start, *event_end));
    printf("%s GPU D2H copy time: %.9f seconds\n", case_name, time_ms / 1000);
    *total_time_ms += time_ms;
}

void init_C_gpu(double *h_C, double *d_C, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_C[i * N + j] = -1.0;
        }
    }

    CUDA_CHECK(hipMemset(d_C, 0, N * N * sizeof(double)));
}

int main(int argc, char *argv[])
{
    // Argument parsing
    if (argc != 3)
    {
        printf("Usage: %s <matrix size NxN> <check>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    int check = atoi(argv[2]);

    printf("Matrix size: %d x %d\n", N, N);

    //
    // Memory allocation
    //
    // Host
    size_t bytes = N * N * sizeof(double);
    double *h_A = (double *)malloc(bytes);
    double *h_B = (double *)malloc(bytes);
    double *h_C = (double *)malloc(bytes);
    double *C = (double *)malloc(bytes);

    // Device
    double *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void **)&d_A, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_B, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_C, bytes));
    CUDA_CHECK(hipMemset(d_C, 0, bytes)); // Init d_C to 0

    //
    // Matrices initialization
    //
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            // Row-major
            h_A[i * N + j] = drand48();
            h_B[i * N + j] = drand48();
            h_C[i * N + j] = -1.0;
            C[i * N + j] = -1.0;
        }
    }

    //
    // Sequential
    //
    if (check)
    {
        struct timespec start, end;
        clock_gettime(CLOCK_MONOTONIC, &start);

        matmul_seq(h_A, h_B, C, N);

        clock_gettime(CLOCK_MONOTONIC, &end);
        double elapsed = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1.0e9;
        printf("Sequential elapsed time: %.9f seconds\n", elapsed);
    }
    else
    {
        printf("Sequential and validation deactivated\n");
    }

    //
    // GPU computations
    //
    hipEvent_t event_start, event_end;
    float time_ms = 0.0;
    float total_time_ms = 0.0;
    CUDA_CHECK(hipEventCreate(&event_start));
    CUDA_CHECK(hipEventCreate(&event_end));

    //
    // Naive kernel
    //
    // Copy Host to Device
    copy_A_B_H2D(h_A, h_B, d_A, d_B, bytes, &event_start, &event_end, &total_time_ms, "Naive");

    // TODO
    // Define threads per block and blocks in the grid
	int n_blocks = (N + BLOCKSIZE-1)/BLOCKSIZE;


    CUDA_CHECK(hipEventRecord(event_start));

    // TODO
    // Launch matmul_naive_kernel
	matmul_naive_kernel<<<n_blocks,BLOCKSIZE>>>(d_A, d_B, d_C, N);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(event_end));
    CUDA_CHECK(hipEventSynchronize(event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_end));
    printf("Naive GPU kernel time: %.9f seconds\n", time_ms / 1000);
    total_time_ms += time_ms;
    time_ms = 0.0;

    // Copy Device to Host
    copy_C_D2H(h_C, d_C, bytes, &event_start, &event_end, &total_time_ms, "Naive");

    printf("Naive GPU total time: %.9f seconds\n", total_time_ms / 1000);
    total_time_ms = 0.0;

    // Validate
    if (check)
        validation(h_C, C, N);

    //
    // Shared memory kernel
    //
    init_C_gpu(h_C, d_C, N);
    // Copy Host to Device
    copy_A_B_H2D(h_A, h_B, d_A, d_B, bytes, &event_start, &event_end, &total_time_ms, "Shared");
    
    // Kernel launch
    CUDA_CHECK(hipEventRecord(event_start));
    // TODO
    // Launch matmul_shared_kernel
	matmul_shared_kernel<<<n_blocks,BLOCKSIZE>>>(d_A, d_B, d_C, N);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(event_end));
    CUDA_CHECK(hipEventSynchronize(event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_end));
    printf("Shared GPU kernel time: %.9f seconds\n", time_ms / 1000);
    total_time_ms += time_ms;
    time_ms = 0.0;

    // Copy Device to Host
    copy_C_D2H(h_C, d_C, bytes, &event_start, &event_end, &total_time_ms, "Shared");

    printf("Shared GPU total time: %.9f seconds\n", total_time_ms / 1000);
    total_time_ms = 0.0;

    // Validate
    if (check)
        validation(h_C, C, N);

    //
    // cuBLAS
    //
    init_C_gpu(h_C, d_C, N);
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    // Copy Host to Device
    copy_A_B_H2D(h_A, h_B, d_A, d_B, bytes, &event_start, &event_end, &total_time_ms, "cuBLAS");

    CUDA_CHECK(hipEventRecord(event_start));

    // TODO
    // Call cuBLAS Matrix Multiplication kernel
	double alpha = 1.0;
	double beta = 0.0;
	for(int i = 0; i < 30; i++) {
		hipblasStatus_t st = hipblasDgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
		if(st != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "CUBLAS error at %s:%d: %d\n", __FILE__, __LINE__, st);
        	exit(1);
		}
	}

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(event_end));
    CUDA_CHECK(hipEventSynchronize(event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_end));
    printf("cuBLAS GPU kernel time: %.9f seconds\n", time_ms / 1000);
    total_time_ms += time_ms;
    time_ms = 0.0;

    // Copy Device to Host
    copy_C_D2H(h_C, d_C, bytes, &event_start, &event_end, &total_time_ms, "cuBLAS");

    printf("cuBLAS GPU total time: %.9f seconds\n", total_time_ms / 1000);

    // Validate
    if (check)
        validation(h_C, C, N);

    //
    // Free memory
    //
    // Host
    free(h_A);
    free(h_B);
    free(h_C);
    free(C);

    // Device
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipEventDestroy(event_start));
    CUDA_CHECK(hipEventDestroy(event_end));
    hipblasDestroy(cublas_handle);

    return 0;
}
