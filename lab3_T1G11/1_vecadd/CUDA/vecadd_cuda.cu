#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "hip/hip_runtime.h"

#define BLOCKSIZE 128
#define CUDA_CHECK(func)                                                                              \
	do {                                                                                              \
		hipError_t err = (func);                                                                     \
		if(err != hipSuccess) {                                                                      \
			fprintf(stderr, "CUDA error at %s:%d %s\n", __FILE__,__LINE__,hipGetErrorString(err));   \
			return 1;                                                                                 \
   		}                                                                                             \
	}while(0);



__global__ void vecadd_cuda(double *A, double *B, double *C, const int N){
		int idx = threadIdx.x + blockIdx.x*blockDim.x;
		if(idx >= N) return;
		C[idx] = A[idx] + B[idx];

}

int main(int argc, char* argv[]){

	int N;

	if(argc < 2){
		printf("Usage: %s <vector size N>\n", argv[0]);
		return 1;
	}

	N = atoi(argv[1]);
	int size = N * sizeof(double);

	double *A = (double*)malloc(size);
	double *B = (double*)malloc(size);
	double *C = (double*)malloc(size);

	for(int i=0; i<N; i++){
		A[i] = (double)i;
		B[i] = (double)(2*(N-i));
	}

	double *d_A, *d_B, *d_C;

	CUDA_CHECK(hipMalloc((void**)&d_A, size));
	CUDA_CHECK(hipMalloc((void**)&d_B, size));
	CUDA_CHECK(hipMalloc((void**)&d_C, size));

	hipEvent_t start, stop;
	CUDA_CHECK(hipEventCreate(&start));
	CUDA_CHECK(hipEventCreate(&stop));

	//LOAD DATA ON DEVICE

	float hostToDeviceTime;

	CUDA_CHECK(hipEventRecord(start));
	CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
	CUDA_CHECK(hipEventRecord(stop));
	CUDA_CHECK(hipEventSynchronize(stop));
	CUDA_CHECK(hipEventElapsedTime(&hostToDeviceTime, start, stop));

	//VECTOR ADDITION

	int n_blocks = (N + BLOCKSIZE-1)/BLOCKSIZE;

	float vecAddTime;

	CUDA_CHECK(hipEventRecord(start));
	vecadd_cuda<<<n_blocks,BLOCKSIZE>>>(d_A, d_B, d_C,N);
	CUDA_CHECK(hipEventRecord(stop));
	CUDA_CHECK(hipEventSynchronize(stop));
	CUDA_CHECK(hipEventElapsedTime(&vecAddTime, start, stop));

	//COPY BACK TO HOST

	float deviceToHostTime;

	CUDA_CHECK(hipEventRecord(start));
	CUDA_CHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipEventRecord(stop));
	CUDA_CHECK(hipEventSynchronize(stop));
	CUDA_CHECK(hipEventElapsedTime(&deviceToHostTime, start, stop));

	for(int i=0; i<N; i++){
		double diff = C[i] - (double)(2*N-i);
		if(fabs(diff) > 1E-6){
			printf("Value exceeding tolerance at i=%d : %lf; diff=%lf\n",i,C[i],diff);
			break;
		}
	}

	//CLEANUP
	free(A); free(B); free(C);
	CUDA_CHECK(hipFree(d_A)); CUDA_CHECK(hipFree(d_B)); CUDA_CHECK(hipFree(d_C));
	CUDA_CHECK(hipEventDestroy(start)); CUDA_CHECK(hipEventDestroy(stop));

	float totalTime = hostToDeviceTime + vecAddTime + deviceToHostTime;

	printf("Vector size: %d\n",N);
	printf("Copy A and B Host to Device elapsed time: %f seconds\n", hostToDeviceTime/1000.0f);
	printf("Kernel elapsed time: %f seconds\n", vecAddTime/1000.0f);
	printf("Copy C Device to Host elapsed time: %f seconds\n", deviceToHostTime/1000.0f);
	printf("Total elapsed time: %f seconds\n", totalTime/1000.0f);
}
